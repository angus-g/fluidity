#include "hip/hip_runtime.h"
/*
  Copyright (c) 2006-2015, The Regents of the University of California,
  through Lawrence Berkeley National Laboratory (subject to receipt of any
  required approvals from the U.S. Dept. of Energy) and the Paul Scherrer
  Institut (Switzerland).  All rights reserved.

  License: see file COPYING in top level of source distribution.
*/

#include "H5hut.h"
#include "examples.h"

#include <stdlib.h>

#include "hip/hip_runtime.h"
// name of input file
const char* fname = "example_setnparticles.h5";

// H5hut verbosity level
const h5_int64_t h5_verbosity = H5_VERBOSE_DEFAULT;

#define USE_CUDA_KERNEL 1
#ifdef USE_CUDA_KERNEL
__global__
#endif
void kernel(h5_int32_t *data, h5_size_t n)
{
	for (h5_size_t i=0; i<n; i++) {
                data[i] += 2;
	}
}


int
main (
        int argc, char* argv[]
        ){

        // initialize MPI & H5hut
        MPI_Init (&argc, &argv);
        MPI_Comm comm = MPI_COMM_WORLD;
        int comm_size = 1;
        MPI_Comm_size (comm, &comm_size);
        int comm_rank = 0;
        MPI_Comm_rank (comm, &comm_rank);
        H5AbortOnError ();
        H5SetVerbosityLevel (h5_verbosity);

        // open file and go to first step
        h5_file_t file = H5OpenFile (fname, H5_O_RDONLY, H5_PROP_DEFAULT);
        H5SetStep (file, 0);

        // compute number of particles this process has to read
        h5_ssize_t num_particles_total = H5PartGetNumParticles (file);
        h5_ssize_t num_particles = num_particles_total / comm_size;
        if (comm_rank+1 == comm_size)
                num_particles += num_particles_total % comm_size;

	printf ("[proc %d]: particles in view: %lld\n", comm_rank, (long long)num_particles);
	printf ("[proc %d]: total number of particles: %lld\n",
		comm_rank, (long long unsigned)num_particles_total);

	// set number of particles
        H5PartSetNumParticles (file, num_particles);

        // read and print data
	h5_int32_t *data;
#ifdef USE_CUDA_KERNEL
	hipMallocManaged((void **)&data, num_particles * sizeof(*data));
#else
	data = (h5_int32_t*)calloc (num_particles, sizeof (*data));
#endif
	H5PartReadDataInt32 (file, "data", data);
        H5CloseFile (file);

#ifdef USE_CUDA_KERNEL
	kernel<<<1, 1>>>(data, num_particles);
#else
	kernel(data, num_particles);
#endif
	int ec=hipDeviceSynchronize();
	printf("%d\n", ec);
	for (int i = 0; i < num_particles; i++) {
                printf ("[proc %d]: local index = %d, value = %d\n",
                        comm_rank, i, data[i]);
        }

        // cleanup
#ifdef USE_CUDA_KERNEL
	hipFree(data);
#else
	free (data);
#endif
	MPI_Finalize ();
        return 0;
}
